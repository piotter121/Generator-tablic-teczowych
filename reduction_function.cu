#include "hip/hip_runtime.h"
#include "reduction_function.h"

__global__ void reduction(char **pass, char **hash) {
	int index = threadIdx.x;
	int i, j = 0;
	//printf("Działania w wątku nr %d\n", index);
	//printf("Otrzymane dane to pass[%d] = %s i hash[%d] = %s\n", index, pass[index], index, hash[index]);
	for (i = 0; i < PASS_LENGTH; i++) {
		pass[index][i] = (hash[index][j] % 10) + '0';
		j = (j + 5) % HASH_LEN; 
	}
	//printf("Po redukcji hasha pass[%d] = %s\n", index, pass[index]);
}

void reduct_rows(char **pass, char **hash, int n) {
	char **d_pass, **d_hash;
	char **d_pass2, **d_hash2;
	int i;	

	d_pass = (char **) malloc(n * sizeof(char *));
	d_hash = (char **) malloc(n * sizeof(char *));

	// printf("Przygotowywanie pamęci na karcie graficznej\n");
	// prepare memory on device
	// printf("Malloc dla d_pass2 \n");
	hipMalloc((void **) &d_pass2, n * sizeof(char *));
	// printf("Malloc dla d_hash2 \n");
	hipMalloc((void **) &d_hash2, n * sizeof(char *));
	
	hipMemcpy(d_pass, d_pass2, n * sizeof(char *), hipMemcpyDeviceToHost);
	hipMemcpy(d_hash, d_hash2, n * sizeof(char *), hipMemcpyDeviceToHost);
	
	for (i = 0; i < n; i++) {
		// printf("Malloc dla d_pass[%d]\n", i);
		hipMalloc((void **) &d_pass[i], PASS_LENGTH * sizeof(char));
		// printf("Malloc dla d_hash[%d]\n", i);
		hipMalloc((void **) &d_hash[i], HASH_LEN * sizeof(char));
		// printf("Kopiowanie pass[%d] = %s i hash[%d] = %s \n", i, pass[i], i, hash[i]);
		hipMemcpy(d_pass[i], pass[i], PASS_LENGTH * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_hash[i], hash[i], HASH_LEN * sizeof(char), hipMemcpyHostToDevice);	
	}
	
	// launch device function
	// printf("Uruchamianie funkcji redukcji na karcie graficznej z %d wątkami\n", n);
	reduction<<<1,n>>>(d_pass, d_hash);

	// copy memory from device to host
	for (i = 0; i < n; i++) {
		// printf("Kopiowanie danych z karty graficznej do pamięci RAM\n");
		hipMemcpy(pass[i], d_pass[i], PASS_LENGTH * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(hash[i], d_hash[i], HASH_LEN * sizeof(char), hipMemcpyDeviceToHost);
		// printf("Odebrane wartości z karty to pass[%d] = %s i hash[%d] = %s\n", i, pass[i], i, hash[i]);
	}

	// free memory
	for (i = 0; i < n; i++) {
		hipFree(d_pass[i]);
		hipFree(d_hash[i]);
	}
	hipFree(d_pass2);
	hipFree(d_hash2);
	free(d_pass);
	free(d_hash);
}


