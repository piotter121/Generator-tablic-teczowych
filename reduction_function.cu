#include "hip/hip_runtime.h"
#include "reduction_function.h"

__global__ void reduction(char **pass, char **hash) {
	int index = threadIdx.x;
	int i, j = 0;
	for (i = 0; i < PASS_LENGTH; i++) {
		pass[index][i] = (hash[index][j] % 10) + '0';
		j = (j + 5) % HASH_LEN; 
	}
}

void reduct_rows(char **pass, char **hash, int n) {
	char **d_pass, **d_hash;
	int i;	

	// prepare memory on device
	hipMalloc((void **) &d_pass, n * sizeof(char *));
	hipMalloc((void **) &d_hash, n * sizeof(char *));
	for (i = 0; i < n; i++) {
		hipMalloc((void **) &d_pass[i], PASS_LENGTH * sizeof(char));
		hipMalloc((void **) &d_hash[i], HASH_LEN * sizeof(char));
		hipMemcpy(d_pass[i], pass[i], PASS_LENGTH * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_hash[i], hash[i], HASH_LEN * sizeof(char), hipMemcpyHostToDevice);	
	}
	
	// launch device function
	reduction<<<1,n>>>(d_pass, d_hash);

	// copy memory from device to host
	for (i = 0; i < n; i++) {
		hipMemcpy(pass[i], d_pass[i], PASS_LENGTH * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(hash[i], d_hash[i], HASH_LEN * sizeof(char), hipMemcpyDeviceToHost);
	}

	// free memory
	hipFree(d_pass);
	hipFree(d_hash);
}


