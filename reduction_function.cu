#include "hip/hip_runtime.h"
#include "reduction_function.h"

__global__ void reduction(char **pass, char **hash) {
	int index = threadIdx.x;
	int i, j = 0;
	for (i = 0; i < PASS_LENGTH; i++) {
		pass[index][i] = (hash[index][j] % 10) + '0';
		j = (j + 5) % HASH_LEN; 
	}
}

void reduct_rows(char **pass, char **hash, int n) {
	char **d_pass, **d_hash;
	char **d_pass2, **d_hash2;
	int i;	

	d_pass = (char **) malloc(n * sizeof(char *));
	d_hash = (char **) malloc(n * sizeof(char *));

	hipMalloc((void **) &d_pass2, n * sizeof(char *));
	hipMalloc((void **) &d_hash2, n * sizeof(char *));
	
	hipMemcpy(d_pass, d_pass2, n * sizeof(char *), hipMemcpyDeviceToHost);
	hipMemcpy(d_hash, d_hash2, n * sizeof(char *), hipMemcpyDeviceToHost);
	
	for (i = 0; i < n; i++) {
		hipMalloc((void **) &d_pass[i], PASS_LENGTH * sizeof(char));
		hipMalloc((void **) &d_hash[i], HASH_LEN * sizeof(char));
		hipMemcpy(d_pass[i], pass[i], PASS_LENGTH * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_hash[i], hash[i], HASH_LEN * sizeof(char), hipMemcpyHostToDevice);	
	}
	
	reduction<<<1,n>>>(d_pass, d_hash);

	for (i = 0; i < n; i++) {
		hipMemcpy(pass[i], d_pass[i], PASS_LENGTH * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(hash[i], d_hash[i], HASH_LEN * sizeof(char), hipMemcpyDeviceToHost);
	}

	for (i = 0; i < n; i++) {
		hipFree(d_pass[i]);
		hipFree(d_hash[i]);
	}
	hipFree(d_pass2);
	hipFree(d_hash2);
	free(d_pass);
	free(d_hash);
}


