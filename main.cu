#include <stdio.h>
#include <stdlib.h>
#include <rpc/des_crypt.h>

#include "table_row.h"

#define SEED 42

void initRow(table_row *row) {
	int i;
	for(i = 0; i < PASS_LENGTH; i++) {
		(*row).first_pass[i] = 97 + rand() % 26;
	}
	(*row).rounds = 0;	
}

int main(int argc, char **argv) {
	int rounds = 5, *rounds_d;
	int nrows = 0, *nrows_d;
	table_row *rows, *rows_d;
	int table_size, i;

	if (argc != 2) {
		printf("Zla liczba argumentow\n");
		exit(EXIT_FAILURE);
	}
	srand(SEED);
	nrows = atoi(argv[1]);
	table_size = nrows * sizeof(table_row);
	rows = (table_row *) malloc(table_size);

	//hipMalloc((void **) &nrows_d, sizeof(int));
	//hipMemcpy(nrows_d, &nrows, sizeof(int), hipMemcpyHostToDevice);
	//hipMalloc((void **) &rounds_d, sizeof(int));
	//hipMemcpy(rounds_d, &rounds, sizeof(int), hipMemcpyHostToDevice);
	//hipMalloc((void **) &rows_d, table_size);

	for (i = 0; i < nrows; i ++) {
		initRow(&rows[i]);
	}
	//hipDeviceSynchronize();

	//hipMemcpy(rows, rows_d, table_size, hipMemcpyDeviceToHost);
	for (i = 0; i < nrows; i++) printf("%s\n", rows[i].first_pass);

	//hipFree(nrows_d);
	//hipFree(rounds_d);
	//hipFree(rows_d);	

	return 0;
}
